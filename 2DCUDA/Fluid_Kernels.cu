#include "hip/hip_runtime.h"
#pragma once
#include "Fluid_Kernels.cuh"

#define index(i,j) ((i) + (DIM) *(j))
#define SWAP(a0, a) {float *tmp = a0; a0 = a; a = tmp;}
#define CLAMP(v, a, b) (a + (v - a) / (b - a))

extern uchar4 *d_textureBufferData;

//velocity and pressure
float *d_u, *d_v;
float *d_u0, *d_v0;
float *d_curl;
//divergence of velocity
float *d_div;

//density
float *d_d, *d_d0;

__global__ void addSource_K(int size, float *d, float *s, float dt) {
	int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	int i = gtidx % size;
	int j = gtidx / size;
	int N = (size - 2);
	
	// Skip Boundary values
	if (i<1 || i>N || j<1 || j>N) return;
	// Add source each timestep
	d[gtidx] += dt * s[gtidx];
}

__global__ void texture_K(int size, uchar4 *surface, float *dens)
{
	int gtidx = (int) (threadIdx.x + blockIdx.x * blockDim.x);
	int i = gtidx % size;
	int j = gtidx / size;
	int N = (size - 2);

	const float treshold1 = 1.;
	const float treshold2 = 4.;
	const float treshold3 = 10.;

	// Skip Boundary values
	if (i<1 || i>N || j<1 || j>N) {
		surface[index(i, j)].w = 255;
		surface[index(i, j)].x = 0;
		surface[index(i, j)].y = 0;
		surface[index(i, j)].z = 0;
		return;
	}
	else
	{
		float pvalue = dens[index(i, j)];
		uchar4 color;
		/* red */
		if (pvalue < treshold1) {
			color.w = 255;
			color.x = 255 * CLAMP(pvalue, 0., treshold1);
			color.y = 0;
			color.z = 0;
		}
		/* yellow */
		else if (pvalue < treshold2) {
			color.w = 255;
			color.x = 255;
			color.y = 255 * (CLAMP(pvalue, treshold1, treshold2) - treshold1);
			color.z = 0;
		}
		/* white */
		else if (pvalue < treshold3){
			color.w = 255;
			color.x = 255;
			color.y = 255;
			color.z = 255 * (CLAMP(pvalue, treshold2, treshold3) - treshold2);
		}
		else{
			color.w = 255;
			color.x = 255;
			color.y = 255;
			color.z = 255;
		}

		if (pvalue > 0) {
			// populate it
			surface[index(i, j)].w = color.w;
			surface[index(i, j)].x = color.x;
			surface[index(i, j)].y = color.y;
			surface[index(i, j)].z = color.z;
		}
		else {
			surface[index(i, j)].w = 255;
			surface[index(i, j)].x = 0;
			surface[index(i, j)].y = 0;
			surface[index(i, j)].z = 0;
		}
	}
}


__global__ void addConstantSource_K(int size, float* x, int i, int j, float value, float dt)
{
	int N = (size - 2);

	// Skip Boundary values
	if (i<1 || i>N || j<1 || j>N) return;

	x[index(i, j)] += value *dt;

	x[index(i+1, j)] += value *dt;
	x[index(i-1, j)] += value *dt;
	x[index(i + 2, j)] += value *dt;
	x[index(i - 2, j)] += value *dt;
}

__global__ void advect_K(int size, float *d, float *d0, float *u, float *v, float dt) {
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;
	int N = (size - 2);
	
	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1, dt0;

	float dx = 1.0f / N;
	dx = 1 / dx;
	dt0 = (dt*dx)/1;
	if (i<1 || i>N || j<1 || j>N) return;

	x = i - dt0*u[index(i, j)];
	y = j - dt0*v[index(i, j)];

	if (x<0.5) x = 0.5;
	if (x>N + 0.5) x = N + 0.5;

	i0 = (int)x;
	i1 = i0 + 1;

	if (y<0.5) y = 0.5;
	if (y>N + 0.5) y = N + 0.5;

	j0 = (int)y;
	j1 = j0 + 1;

	s1 = x - i0;
	s0 = 1 - s1;
	t1 = y - j0;
	t0 = 1 - t1;
	d[index(i, j)] = s0 * (t0*d0[index(i0, j0)] + t1*d0[index(i0, j1)]) +
		s1 * (t0*d0[index(i1, j0)] + t1*d0[index(i1, j1)]);
}

__global__ void redGauss_K(int size, float *x, float *x0, float a, float c)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;
	float invC = 1.f / c;
	int N = (size - 2);

	if (i<1 || i>N || j<1 || j>N) return;

	if ((i + j) % 2 == 0)
	{
		x[index(i, j)] = (x0[index(i, j)] +	a * (x[index(i - 1, j)] + x[index(i + 1, j)] + x[index(i, j - 1)] +	x[index(i, j + 1)])) * invC;
	}
}

__global__ void blackGauss_K(int size, float *x, float *x0, float a, float c)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;
	float invC = 1.f / c;
	int N = (size - 2);

	if (i<1 || i>N || j<1 || j>N) return;

	if ((i + j) % 2 != 0)
	{
		x[index(i, j)] = (x0[index(i, j)] +	a * (x[index(i - 1, j)] + x[index(i + 1, j)] + x[index(i, j - 1)] +	x[index(i, j + 1)])) * invC;
	}
}

__global__ void divergence_K(int size, float* u, float* v, float* p, float* div) {
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;

	int N = (size - 2);

	if (i<1 || i>N || j<1 || j>N) return;
	
		float h = 1.0f / N;
		// Calculate divergence using finite difference method
		// We multiply by -1 here to reduce the number of negative multiplications in the pressure calculation
		div[index(i, j)] = -0.5f*h*(u[index(i + 1, j)] - u[index(i - 1, j)] + v[index(i, j + 1)] - v[index(i, j - 1)]);
		p[index(i, j)] = 0;
	
}

__global__ void subtractGradient_K(int size, float *u, float *v, float *p)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;

	// Skip Boundary values

	int N = (size - 2);

	if (i<1 || i>N || j<1 || j>N) return;
		float h = 1.0f / N;
		// Calculate divergence using finite difference method
		// We multiply by -1 here to reduce the number of negative multiplications in the pressure calculation
		u[index(i, j)] -= 0.5*(p[index(i + 1, j)] - p[index(i - 1, j)]) / h;
		v[index(i, j)] -= 0.5*(p[index(i, j + 1)] - p[index(i, j - 1)]) / h;
	
}

__global__ void curl_K(int size, float *u, float *v, float *curl)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;

	// Skip Boundary values
	int N = (size - 2);
	if (i<1 || i>N || j<1 || j>N) return;

	float h = 1.0f / N;
	float du_dy;
	float dv_dx;

	du_dy = (u[index(i, j + 1)] - u[index(i, j - 1)]) /h * 0.5f;
	dv_dx = (v[index(i + 1, j)] - v[index(i - 1, j)]) /h * 0.5f;

	curl[index(i, j)] = (dv_dx - du_dy);
}

__global__ void vorticity_K(int size, float *u, float *v, float *curl, float vort_str, float dt)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx % size;
	int j = gtidx / size;

	// Skip Boundary values
	int N = (size - 2);
	if (i<1 || i>N || j<1 || j>N) return;

	float h = 1.0f / N;

	float vort;

	float omegaT = curl[index(i, j - 1)];
	float omegaB = curl[index(i, j + 1)];
	float omegaR = curl[index(i + 1, j)];
	float omegaL = curl[index(i - 1, j)];

	float dw_dx = (omegaR - omegaL) * 0.5f;
	float dw_dy = (omegaT - omegaB) * 0.5f;
	float2 force; force.x = dw_dy; force.y = dw_dx; force /= h;
	force /= (length(force) + 0.000001f);
	
	float2 newVec;
	newVec.x = -curl[index(i, j)] * force.y;
	newVec.y = curl[index(i, j)] * force.x;
	newVec *= vort_str;
	u[index(i, j)] += newVec.x * dt;
	v[index(i, j)] += newVec.y * dt;

}

__global__ void buoyancy_K(int size, float *d, float *s, float kappa, float sigma)
{
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = (int)gtidx % size;
	int j = (int)gtidx / size;
	int N = (size - 2);

	// Skip Boundary values
	if (i<1 || i>N || j<1 || j>N) return;

	d[index(i, j)] = sigma*s[index(i, j)] + -kappa* s[index(i, j)];
}

__global__ void set_bnd_K(int size, int b, float *x) {
	int gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = gtidx + 1;
	int N = size-2;

	if (i <= N){
		x[index(0, i)] = b == 1 ? -x[index(1, i)] : x[index(1, i)];
		x[index(N + 1, i)] = b == 1 ? -x[index(N, i)] : x[index(N, i)];
		x[index(i, 0)] = b == 2 ? -x[index(i, 1)] : x[index(i, 1)];
		x[index(i, N + 1)] = b == 2 ? -x[index(i, N)] : x[index(i, N)];

		if (i == 1)
		{
			x[index(0, 0)] = 0.5f*(x[index(1, 0)] + x[index(0, 1)]);
			x[index(0, N + 1)] = 0.5f*(x[index(1, N + 1)] + x[index(0, N)]);
			x[index(N + 1, 0)] = 0.5f*(x[index(N, 0)] + x[index(N + 1, 1)]);
			x[index(N + 1, N + 1)] = 0.5f*(x[index(N, N + 1)] + x[index(N + 1, N)]);
		}
	}
}


extern "C"
void initCUDA(int dim)
{
	hipSetDevice(0);
	hipMalloc((void**)&d_div, dim * sizeof(float));
	hipMalloc((void**)&d_d, dim * sizeof(float));
	hipMalloc((void**)&d_d0, dim * sizeof(float));
	hipMalloc((void**)&d_u, dim * sizeof(float));
	hipMalloc((void**)&d_u0, dim * sizeof(float));
	hipMalloc((void**)&d_v, dim * sizeof(float));
	hipMalloc((void**)&d_v0, dim * sizeof(float));
	hipMalloc((void**)&d_curl, dim * sizeof(float));

	// Initialize our "previous" values of density and velocity to be all zero
	hipMemset(d_u, 0, dim * sizeof(float));
	hipMemset(d_v, 0, dim * sizeof(float));
	hipMemset(d_d, 0, dim * sizeof(float));
	hipMemset(d_u0, 0, dim * sizeof(float));
	hipMemset(d_v0, 0, dim * sizeof(float));
	hipMemset(d_d0, 0, dim * sizeof(float));
	hipMemset(d_div, 0, dim * sizeof(float));
	hipMemset(d_curl, 0, dim * sizeof(float));
}

extern "C"
void freeCUDA()
{
	hipFree(d_d);
	hipFree(d_d0);
	hipFree(d_u);
	hipFree(d_u0);
	hipFree(d_v);
	hipFree(d_v0);
	hipFree(d_div);
	hipFree(d_curl);
	hipFree(d_textureBufferData);
	hipDeviceReset();
}

void diffuse(int size, int b, float *x, float *x0, float diff, int iteration)
{
	int N = (size - 2);
	float a = 0.01f * diff * (float) N * (float) N;
	float c = 1.f + 4.f *a;
	for (int i = 0; i < iteration; i++)
	{
		redGauss_K<<<BLOCKS, THREADS>>>(size, x, x0, a, c);
		hipDeviceSynchronize();
		blackGauss_K<<<BLOCKS, THREADS>>>(size, x, x0, a, c);
	}

	hipDeviceSynchronize();
	set_bnd_K<<<1, N>>>(size, b, x);
	hipDeviceSynchronize();
}

void advect(int size, int b, float *d, float *d0, float *u, float *v, float dt)
{
	int N = (size - 2);

	advect_K<<<BLOCKS, THREADS>>>(size, d, d0, u, v, dt);
	hipDeviceSynchronize();
	set_bnd_K<<<1, N >>>(size, b, d);
	hipDeviceSynchronize();
}

void project(int size, float *u, float *v, float *p, float *div, int iteration)
{
	int N = (size - 2);

	divergence_K<<<BLOCKS, THREADS>>>(size, u, v, p, div);
	hipDeviceSynchronize();
	set_bnd_K<<<1, N >>>(size, 0, div);
	set_bnd_K<<<1, N >>>(size, 0, p);
	hipDeviceSynchronize();

	for (int k = 0; k < iteration; k++){
		// Linear Solve
		redGauss_K<<<BLOCKS, THREADS>>>(size, p, div, 1, 4);
		hipDeviceSynchronize();
		blackGauss_K<<<BLOCKS, THREADS>>>(size, p, div, 1, 4);
		hipDeviceSynchronize();
		set_bnd_K<<<1, N>>>(size, 0, p);
		hipDeviceSynchronize();
	}

	subtractGradient_K<<<BLOCKS, THREADS>>>(size, u, v, p);
	hipDeviceSynchronize();
	set_bnd_K<<<1, N >>>(size, 1, u);
	set_bnd_K<<<1, N >>>(size, 2, v);
	hipDeviceSynchronize();
}

extern "C"
void step(int size,
		  float dt, 
		  float viscosity, 
		  float diffusion, 
		  float kappa, 
		  float sigma, 
		  int iteration,
		  float *sd,
		  float s_v_i,
		  float s_v_j,
		  float s_d_i,
		  float s_d_j,
		  float s_d_val,
		  float s_u_val,
		  float s_v_val)
{
	int N = (size - 2);
	// Vel step
	// Add Velocity Source
	addConstantSource_K<<<1, 1>>>(size, d_u, s_v_i, s_v_j, s_u_val, dt);
	addConstantSource_K<<<1, 1>>>(size, d_v, s_v_i, s_v_j, s_v_val, dt);
	hipDeviceSynchronize();

	//Vorticity
	if (true) {
		curl_K<<<BLOCKS, THREADS>>>(size, d_u, d_v, d_curl);
		hipDeviceSynchronize();
		vorticity_K<<<BLOCKS, THREADS>>>(size, d_u, d_v, d_curl, 0.10f, dt);
		hipDeviceSynchronize();
		set_bnd_K<<<1, N >>>(size, 1, d_u);
		set_bnd_K<<<1, N >>>(size, 2, d_v);
		hipDeviceSynchronize();
	}

	// Buoyancy
	if (true) {
		buoyancy_K<<<BLOCKS, THREADS>>>(size, d_v0, d_d, kappa, sigma);
		hipDeviceSynchronize();
		addSource_K<<<BLOCKS, THREADS>>>(size, d_v, d_v0, dt);
		hipDeviceSynchronize();
		set_bnd_K<<<1, N>>>(size, 2, d_v);
		hipDeviceSynchronize();
	}
	
	SWAP(d_u0, d_u);
	diffuse(size, 1, d_u, d_u0, viscosity, iteration);
	SWAP(d_v0, d_v);
	diffuse(size, 2, d_v, d_v0, viscosity, iteration);

	project(size, d_u, d_v, d_u0, d_v0, iteration);

	SWAP(d_u0, d_u);
	SWAP(d_v0, d_v);
	advect(size, 1, d_u, d_u0, d_u0, d_v0, dt);
	advect(size, 1, d_v, d_v0, d_u0, d_v0, dt);

	project(size, d_u, d_v, d_u0, d_v0, iteration);
	

	// Density step
	// Add Density Source
	addConstantSource_K<<<1, 1>>>(size, d_d, s_d_i, s_d_j, s_d_val, dt);
	addConstantSource_K<<<1, 1>>>(size, d_d, 128, 248, 100, dt);
	hipDeviceSynchronize();

	SWAP(d_d0, d_d);
	diffuse(size, 0, d_d, d_d0, diffusion, iteration);
	SWAP(d_d0, d_d);
	advect(size, 0, d_d, d_d0, d_u, d_v, dt);

	//hipMemcpy(sd, d_d, (size*size)*sizeof(float), hipMemcpyDeviceToHost);

	// Reset for next step
	hipMemset(d_u0, 0, (size*size) * sizeof(float));
	hipMemset(d_v0, 0, (size*size) * sizeof(float));
	hipMemset(d_d0, 0, (size*size) * sizeof(float));
	return;
}

extern "C"
void createTexture(int size, uchar4* d_texture)
{
	texture_K<<<BLOCKS,THREADS>>>(size, d_texture, d_d);
}